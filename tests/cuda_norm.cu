#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_fp16.h>
#include <mateval/cuda/norm.hpp>

namespace {
template <class T>
const char* get_name();
template <> const char* get_name<half  >() {return "half"  ;}
template <> const char* get_name<float >() {return "float" ;}
template <> const char* get_name<double>() {return "double";}
} // noname namespace

template <class T>
void eval(
	const std::size_t len
	) {
	T* array_ptr;
	hipMallocManaged(&array_ptr, sizeof(T) * len);
	for (std::size_t i = 0; i < len; i++) {
		array_ptr[i] = static_cast<double>(i + 1);
	}

	const auto norm = mtk::mateval::cuda::norm(array_ptr, len);
	const auto ref_norm = std::sqrt(len * (len + 1) * (2 * len + 1) / 6);

	std::printf("[dtype = %6s, len = %lu] norm = %e, ref = %e, %s\n",
							get_name<T>(), len,
							norm, ref_norm,
							std::abs(norm - ref_norm) < ref_norm * 1e-4 ? "OK" : "NG");
}

int main() {
	eval<half  >(1lu << 10);
	eval<float >(1lu << 10);
	eval<double>(1lu << 10);
	eval<half  >(1lu << 12);
	eval<float >(1lu << 20);
	eval<double>(1lu << 20);
}
